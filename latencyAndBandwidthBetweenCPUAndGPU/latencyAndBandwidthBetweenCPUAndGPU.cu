#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for pageable
 * and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization

#include <hip/hip_runtime.h>

#include <memory>
#include <iostream>
#include <cassert>

#include <cstdio>
#include <vector>

static const char *sSDKsample = "Latency and Bandwidth Test";

// defines, project
#define MEMCOPY_ITERATIONS  100
#define MEMCOPY_ITERATIONS_For_Test_Latency  100
#define DEFAULT_SIZE        ( 32 * ( 1 << 20 ) )    //32 M
#define DEFAULT_SIZE_For_Test_Latency        ( 1 )
#define DEFAULT_INCREMENT   (1 << 22)               //4 M
#define CACHE_CLEAR_SIZE    (1 << 24)               //16 M

//shmoo mode defines
#define SHMOO_MEMSIZE_MAX     (1 << 26)         //64 M
#define SHMOO_MEMSIZE_START   (1 << 10)         //1 KB
#define SHMOO_INCREMENT_1KB   (1 << 10)         //1 KB
#define SHMOO_INCREMENT_2KB   (1 << 11)         //2 KB
#define SHMOO_INCREMENT_10KB  (10 * (1 << 10))  //10KB
#define SHMOO_INCREMENT_100KB (100 * (1 << 10)) //100 KB
#define SHMOO_INCREMENT_1MB   (1 << 20)         //1 MB
#define SHMOO_INCREMENT_2MB   (1 << 21)         //2 MB
#define SHMOO_INCREMENT_4MB   (1 << 22)         //4 MB
#define SHMOO_LIMIT_20KB      (20 * (1 << 10))  //20 KB
#define SHMOO_LIMIT_50KB      (50 * (1 << 10))  //50 KB
#define SHMOO_LIMIT_100KB     (100 * (1 << 10)) //100 KB
#define SHMOO_LIMIT_1MB       (1 << 20)         //1 MB
#define SHMOO_LIMIT_16MB      (1 << 24)         //16 MB
#define SHMOO_LIMIT_32MB      (1 << 25)         //32 MB

//enums, project
enum testMode   { QUICK_MODE, RANGE_MODE, SHMOO_MODE };
enum memcpyKind { DEVICE_TO_HOST, HOST_TO_DEVICE, DEVICE_TO_DEVICE };
enum printMode  { USER_READABLE, CSV };
enum memoryMode { PINNED, PAGEABLE };

const char *sMemoryCopyKind[] =
{
    "Device to Host",
    "Host to Device",
    "Device to Device",
    NULL
};

const char *sMemoryMode[] =
{
    "PINNED",
    "PAGEABLE",
    NULL
};

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

int *pArgc = NULL;
char **pArgv = NULL;

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runBandwidthTest(const int argc, const char **argv);
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthRange(unsigned int start, unsigned int end, unsigned int increment,
                        memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthShmoo(memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testDeviceToDeviceTransfer(unsigned int memSize);
void outputLatencyMatrix(int numGPUs, bool p2p, memcpyKind kind);
void printResultsReadable(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc);
void printResultsCSV(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc);
void printHelp(void);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    pArgc = &argc;
    pArgv = argv;

    // set logfile name and start logs
    printf("[%s] - Starting...\n", sSDKsample);

    int iRetVal = runBandwidthTest(argc, (const char **)argv);

    if (iRetVal < 0)
    {
        checkCudaErrors(hipSetDevice(0));
    }

    // latency tests
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }
    printf("\nHOST_TO_DEVICE \n");
    outputLatencyMatrix(deviceCount, true, HOST_TO_DEVICE);

    printf("\nDEVICE_TO_HOST \n");
    outputLatencyMatrix(deviceCount, true, DEVICE_TO_HOST);

    printf("\nDEVICE_TO_DEVICE \n");
    outputLatencyMatrix(deviceCount, true, DEVICE_TO_DEVICE);

    // finish
    printf("%s\n", (iRetVal==0) ? "Result = PASS" : "Result = FAIL");

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    exit((iRetVal==0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////
//Parse args, run the appropriate tests
///////////////////////////////////////////////////////////////////////////////
int runBandwidthTest(const int argc, const char **argv)
{
    int start = DEFAULT_SIZE;
    int end = DEFAULT_SIZE;
    int startDevice = 0;
    int endDevice = 0;
    int increment = DEFAULT_INCREMENT;
    testMode mode = QUICK_MODE;
    bool htod = false;
    bool dtoh = false;
    bool dtod = false;
    bool wc = false;
    char *modeStr;
    char *device = NULL;
    printMode printmode = USER_READABLE;
    char *memModeStr = NULL;
    memoryMode memMode = PINNED;

    //process command line args
    if (checkCmdLineFlag(argc, argv, "help"))
    {
        printHelp();
        return 0;
    }

    if (checkCmdLineFlag(argc, argv, "csv"))
    {
        printmode = CSV;
    }

    if (getCmdLineArgumentString(argc, argv, "memory", &memModeStr))
    {
        if (strcmp(memModeStr, "pageable") == 0)
        {
            memMode = PAGEABLE;
        }
        else if (strcmp(memModeStr, "pinned") == 0)
        {
            memMode = PINNED;
        }
        else
        {
            printf("Invalid memory mode - valid modes are pageable or pinned\n");
            printf("See --help for more information\n");
            return -1000;
        }
    }
    else
    {
        //default - pinned memory
        memMode = PINNED;
    }

    if (getCmdLineArgumentString(argc, argv, "device", &device))
    {
        int deviceCount;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);

        if (error_id != hipSuccess)
        {
            printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
            exit(EXIT_FAILURE);
        }

        if (deviceCount == 0)
        {
            printf("!!!!!No devices found!!!!!\n");
            return -2000;
        }

        if (strcmp(device, "all") == 0)
        {
            printf("\n!!!!!Cumulative Bandwidth to be computed from all the devices !!!!!!\n\n");
            startDevice = 0;
            endDevice = deviceCount-1;
        }
        else
        {
            startDevice = endDevice = atoi(device);

            if (startDevice >= deviceCount || startDevice < 0)
            {
                printf("\n!!!!!Invalid GPU number %d given hence default gpu %d will be used !!!!!\n", startDevice,0);
                startDevice = endDevice = 0;
            }
        }
    }
    else
    {
    	printf("Device No is not set, use the default device: 0.\n");
    }

    printf("Running Bandwidth ...\n\n");

    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipDeviceProp_t deviceProp;
        hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

        if (error_id == hipSuccess)
        {
            printf(" Device %d: %s\n", currentDevice, deviceProp.name);

            if (deviceProp.computeMode == hipComputeModeProhibited)
            {
                fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
                checkCudaErrors(hipSetDevice(currentDevice));

                exit(EXIT_FAILURE);
            }
        }
        else
        {
            printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
            checkCudaErrors(hipSetDevice(currentDevice));

            exit(EXIT_FAILURE);
        }
    }

    if (getCmdLineArgumentString(argc, argv, "mode", &modeStr))
    {
        //figure out the mode
        if (strcmp(modeStr, "quick") == 0)
        {
            printf(" Quick Mode\n\n");
            mode = QUICK_MODE;
        }
        else if (strcmp(modeStr, "shmoo") == 0)
        {
            printf(" Shmoo Mode\n\n");
            mode = SHMOO_MODE;
        }
        else if (strcmp(modeStr, "range") == 0)
        {
            printf(" Range Mode\n\n");
            mode = RANGE_MODE;
        }
        else
        {
            printf("Invalid mode - valid modes are quick, range, or shmoo\n");
            printf("See --help for more information\n");
            return -3000;
        }
    }
    else
    {
        //default mode - quick
        printf(" Quick Mode\n\n");
        mode = QUICK_MODE;
    }

    if (checkCmdLineFlag(argc, argv, "htod"))
    {
        htod = true;
    }

    if (checkCmdLineFlag(argc, argv, "dtoh"))
    {
        dtoh = true;
    }

    if (checkCmdLineFlag(argc, argv, "dtod"))
    {
        dtod = true;
    }

#if CUDART_VERSION >= 2020

    if (checkCmdLineFlag(argc, argv, "wc"))
    {
        wc = true;
    }

#endif

    if (checkCmdLineFlag(argc, argv, "cputiming"))
    {
        bDontUseGPUTiming = true;
    }

    if (!htod && !dtoh && !dtod)
    {
        //default:  All
        htod = true;
        dtoh = true;
        dtod = true;
    }

    if (RANGE_MODE == mode)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "start"))
        {
            start = getCmdLineArgumentInt(argc, argv, "start");

            if (start <= 0)
            {
                printf("Illegal argument - start must be greater than zero\n");
                return -4000;
            }
        }
        else
        {
            printf("Must specify a starting size in range mode\n");
            printf("See --help for more information\n");
            return -5000;
        }

        if (checkCmdLineFlag(argc, (const char **)argv, "end"))
        {
            end = getCmdLineArgumentInt(argc, argv, "end");

            if (end <= 0)
            {
                printf("Illegal argument - end must be greater than zero\n");
                return -6000;
            }

            if (start > end)
            {
                printf("Illegal argument - start is greater than end\n");
                return -7000;
            }
        }
        else
        {
            printf("Must specify an end size in range mode.\n");
            printf("See --help for more information\n");
            return -8000;
        }


        if (checkCmdLineFlag(argc, argv, "increment"))
        {
            increment = getCmdLineArgumentInt(argc, argv, "increment");

            if (increment <= 0)
            {
                printf("Illegal argument - increment must be greater than zero\n");
                return -9000;
            }
        }
        else
        {
            printf("Must specify an increment in user mode\n");
            printf("See --help for more information\n");
            return -10000;
        }
    }

    if (htod)
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, HOST_TO_DEVICE, printmode, memMode, startDevice, endDevice, wc);
    }

    if (dtoh)
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, DEVICE_TO_HOST, printmode, memMode, startDevice, endDevice, wc);
    }

    if (dtod)
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, DEVICE_TO_DEVICE, printmode, memMode, startDevice, endDevice, wc);
    }

    // Ensure that we reset all CUDA Devices in question
    for (int nDevice = startDevice; nDevice <= endDevice; nDevice++)
    {
        hipSetDevice(nDevice);
    }

    return 0;
}

///////////////////////////////////////////////////////////////////////////////
//  Run a bandwidth test
///////////////////////////////////////////////////////////////////////////////
void
testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
              testMode mode, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    switch (mode)
    {
        case QUICK_MODE:
            testBandwidthQuick(DEFAULT_SIZE, kind, printmode, memMode, startDevice, endDevice, wc);
            break;

        case RANGE_MODE:
            testBandwidthRange(start, end, increment, kind, printmode, memMode, startDevice, endDevice, wc);
            break;

        case SHMOO_MODE:
            testBandwidthShmoo(kind, printmode, memMode, startDevice, endDevice, wc);
            break;

        default:
            break;
    }
}

//////////////////////////////////////////////////////////////////////
//  Run a quick mode bandwidth test
//////////////////////////////////////////////////////////////////////
void
testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    testBandwidthRange(size, size, DEFAULT_INCREMENT, kind, printmode, memMode, startDevice, endDevice, wc);
}

///////////////////////////////////////////////////////////////////////
//  Run a range mode bandwidth test
//////////////////////////////////////////////////////////////////////
void
testBandwidthRange(unsigned int start, unsigned int end, unsigned int increment,
                   memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    //count the number of copies we're going to run
    unsigned int count = 1 + ((end - start) / increment);

    unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
    double *bandwidths = (double *) malloc(count * sizeof(double));

    // Before calculating the cumulative bandwidth, initialize bandwidths array to NULL
    for (unsigned int i = 0; i < count; i++)
    {
        bandwidths[i] = 0.0;
    }

    // Use the device asked by the user
    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipSetDevice(currentDevice);

        //run each of the copies
        for (unsigned int i = 0; i < count; i++)
        {

            memSizes[i] = start + i * increment;

            switch (kind)
            {
                case DEVICE_TO_HOST:
                    bandwidths[i] += testDeviceToHostTransfer(memSizes[i], memMode, wc);
                    break;

                case HOST_TO_DEVICE:
                    bandwidths[i] += testHostToDeviceTransfer(memSizes[i], memMode, wc);
                    break;

                case DEVICE_TO_DEVICE:
                    bandwidths[i] += testDeviceToDeviceTransfer(memSizes[i]);
                    break;
            }
        }
    } // Complete the bandwidth computation on all the devices

    //print results
    if (printmode == CSV)
    {
        printResultsCSV(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }
    else
    {
        printResultsReadable(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }

    //clean up
    free(memSizes);
    free(bandwidths);
}

//////////////////////////////////////////////////////////////////////////////
// Intense shmoo mode - covers a large range of values with varying increments
//////////////////////////////////////////////////////////////////////////////
void
testBandwidthShmoo(memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    //count the number of copies to make
    unsigned int count = 1 + (SHMOO_LIMIT_20KB  / SHMOO_INCREMENT_1KB)
                         + ((SHMOO_LIMIT_50KB - SHMOO_LIMIT_20KB) / SHMOO_INCREMENT_2KB)
                         + ((SHMOO_LIMIT_100KB - SHMOO_LIMIT_50KB) / SHMOO_INCREMENT_10KB)
                         + ((SHMOO_LIMIT_1MB - SHMOO_LIMIT_100KB) / SHMOO_INCREMENT_100KB)
                         + ((SHMOO_LIMIT_16MB - SHMOO_LIMIT_1MB) / SHMOO_INCREMENT_1MB)
                         + ((SHMOO_LIMIT_32MB - SHMOO_LIMIT_16MB) / SHMOO_INCREMENT_2MB)
                         + ((SHMOO_MEMSIZE_MAX - SHMOO_LIMIT_32MB) / SHMOO_INCREMENT_4MB);

    unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
    double *bandwidths = (double *) malloc(count * sizeof(double));

    // Before calculating the cumulative bandwidth, initialize bandwidths array to NULL
    for (unsigned int i = 0; i < count; i++)
    {
        bandwidths[i] = 0.0;
    }

    // Use the device asked by the user
    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipSetDevice(currentDevice);
        //Run the shmoo
        int iteration = 0;
        unsigned int memSize = 0;

        while (memSize <= SHMOO_MEMSIZE_MAX)
        {
            if (memSize < SHMOO_LIMIT_20KB)
            {
                memSize += SHMOO_INCREMENT_1KB;
            }
            else if (memSize < SHMOO_LIMIT_50KB)
            {
                memSize += SHMOO_INCREMENT_2KB;
            }
            else if (memSize < SHMOO_LIMIT_100KB)
            {
                memSize += SHMOO_INCREMENT_10KB;
            }
            else if (memSize < SHMOO_LIMIT_1MB)
            {
                memSize += SHMOO_INCREMENT_100KB;
            }
            else if (memSize < SHMOO_LIMIT_16MB)
            {
                memSize += SHMOO_INCREMENT_1MB;
            }
            else if (memSize < SHMOO_LIMIT_32MB)
            {
                memSize += SHMOO_INCREMENT_2MB;
            }
            else
            {
                memSize += SHMOO_INCREMENT_4MB;
            }

            memSizes[iteration] = memSize;

            switch (kind)
            {
                case DEVICE_TO_HOST:
                    bandwidths[iteration] += testDeviceToHostTransfer(memSizes[iteration], memMode, wc);
                    break;

                case HOST_TO_DEVICE:
                    bandwidths[iteration] += testHostToDeviceTransfer(memSizes[iteration], memMode, wc);
                    break;

                case DEVICE_TO_DEVICE:
                    bandwidths[iteration] += testDeviceToDeviceTransfer(memSizes[iteration]);
                    break;
            }

            iteration++;
            printf(".");
        }
    } // Complete the bandwidth computation on all the devices

    //print results
    printf("\n");

    if (CSV == printmode)
    {
        printResultsCSV(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }
    else
    {
        printResultsReadable(memSizes, bandwidths, count, kind, memMode, (1 + endDevice - startDevice), wc);
    }

    //clean up
    free(memSizes);
    free(bandwidths);
}

///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    unsigned char *h_idata = NULL;
    unsigned char *h_odata = NULL;
    hipEvent_t start, stop;

    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //allocate host memory
    if (PINNED == memMode)
    {
        //pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostAlloc((void **)&h_idata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
        checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
#else
        checkCudaErrors(hipHostMalloc((void **)&h_idata, memSize));
        checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
    }
    else
    {
        //pageable memory mode - use malloc
        h_idata = (unsigned char *)malloc(memSize);
        h_odata = (unsigned char *)malloc(memSize);

        if (h_idata == 0 || h_odata == 0)
        {
            fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
            exit(EXIT_FAILURE);
        }
    }

    //initialize the memory
    for (unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_idata[i] = (unsigned char)(i & 0xff);
    }

    // allocate device memory
    unsigned char *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, memSize));

    //initialize the device memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, memSize,
                               hipMemcpyHostToDevice));

    //copy data from GPU to Host
    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    if (PINNED == memMode)
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpyAsync(h_odata, d_idata, memSize,
                                            hipMemcpyDeviceToHost, 0));
        }
    }
    else
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpy(h_odata, d_idata, memSize,
                                       hipMemcpyDeviceToHost));
        }
    }

    checkCudaErrors(hipEventRecord(stop, 0));

    // make sure GPU has finished copying
    checkCudaErrors(hipDeviceSynchronize());
    //get the total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if (PINNED != memMode || bDontUseGPUTiming)
    {
        elapsedTimeInMs = sdkGetTimerValue(&timer);
    }

    //calculate bandwidth in MB/s
    bandwidthInMBs = ((float)(1<<10) * memSize * (float)MEMCOPY_ITERATIONS) /
                     (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    sdkDeleteTimer(&timer);

    if (PINNED == memMode)
    {
        checkCudaErrors(hipHostFree(h_idata));
        checkCudaErrors(hipHostFree(h_odata));
    }
    else
    {
        free(h_idata);
        free(h_odata);
    }

    checkCudaErrors(hipFree(d_idata));

    return bandwidthInMBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;
    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //allocate host memory
    unsigned char *h_odata = NULL;

    if (PINNED == memMode)
    {
#if CUDART_VERSION >= 2020
        //pinned memory mode - use special function to get OS-pinned memory
        checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
#else
        //pinned memory mode - use special function to get OS-pinned memory
        checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
    }
    else
    {
        //pageable memory mode - use malloc
        h_odata = (unsigned char *)malloc(memSize);

        if (h_odata == 0)
        {
            fprintf(stderr, "Not enough memory available on host to run test!\n");
            exit(EXIT_FAILURE);
        }
    }

    unsigned char *h_cacheClear1 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);
    unsigned char *h_cacheClear2 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);

    if (h_cacheClear1 == 0 || h_cacheClear2 == 0)
    {
        fprintf(stderr, "Not enough memory available on host to run test!\n");
        exit(EXIT_FAILURE);
    }

    //initialize the memory
    for (unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_odata[i] = (unsigned char)(i & 0xff);
    }

    for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++)
    {
        h_cacheClear1[i] = (unsigned char)(i & 0xff);
        h_cacheClear2[i] = (unsigned char)(0xff - (i & 0xff));
    }

    //allocate device memory
    unsigned char *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, memSize));

    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    //copy host memory to device memory
    if (PINNED == memMode)
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpyAsync(d_idata, h_odata, memSize,
                                            hipMemcpyHostToDevice, 0));
        }
    }
    else
    {
        for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            checkCudaErrors(hipMemcpy(d_idata, h_odata, memSize,
                                       hipMemcpyHostToDevice));
        }
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    //total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if (PINNED != memMode || bDontUseGPUTiming)
    {
        elapsedTimeInMs = sdkGetTimerValue(&timer);
    }

    sdkResetTimer(&timer);

    //calculate bandwidth in MB/s
    bandwidthInMBs = ((float)(1<<10) * memSize * (float)MEMCOPY_ITERATIONS) /
                     (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    sdkDeleteTimer(&timer);

    if (PINNED == memMode)
    {
        checkCudaErrors(hipHostFree(h_odata));
    }
    else
    {
        free(h_odata);
    }

    free(h_cacheClear1);
    free(h_cacheClear2);
    checkCudaErrors(hipFree(d_idata));

    return bandwidthInMBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testDeviceToDeviceTransfer(unsigned int memSize)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;

    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //allocate host memory
    unsigned char *h_idata = (unsigned char *)malloc(memSize);

    if (h_idata == 0)
    {
        fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
        exit(EXIT_FAILURE);
    }

    //initialize the host memory
    for (unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_idata[i] = (unsigned char)(i & 0xff);
    }

    //allocate device memory
    unsigned char *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, memSize));
    unsigned char *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, memSize));

    //initialize memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, memSize,
                               hipMemcpyHostToDevice));

    //run the memcopy
    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
        checkCudaErrors(hipMemcpy(d_odata, d_idata, memSize,
                                   hipMemcpyDeviceToDevice));
    }

    checkCudaErrors(hipEventRecord(stop, 0));

    //Since device to device memory copies are non-blocking,
    //hipDeviceSynchronize() is required in order to get
    //proper timing.
    checkCudaErrors(hipDeviceSynchronize());

    //get the total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if (bDontUseGPUTiming)
    {
        elapsedTimeInMs = sdkGetTimerValue(&timer);
    }

    //calculate bandwidth in MB/s
    bandwidthInMBs = 2.0f * ((float)(1<<10) * memSize * (float)MEMCOPY_ITERATIONS) /
                     (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    sdkDeleteTimer(&timer);
    free(h_idata);
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return bandwidthInMBs;
}

/////////////////////////////////////////////////////////
//print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc)
{
    printf(" %s Bandwidth, %i Device(s)\n", sMemoryCopyKind[kind], iNumDevs);
    printf(" %s Memory Transfers\n", sMemoryMode[memMode]);

    if (wc)
    {
        printf(" Write-Combined Memory Writes are Enabled");
    }

    printf("   Transfer Size (Bytes)\tBandwidth(MB/s)\n");
    unsigned int i;

    for (i = 0; i < (count - 1); i++)
    {
        printf("   %u\t\t\t%s%.1f\n", memSizes[i], (memSizes[i] < 10000)? "\t" : "", bandwidths[i]);
    }

    printf("   %u\t\t\t%s%.1f\n\n", memSizes[i], (memSizes[i] < 10000)? "\t" : "", bandwidths[i]);
}

///////////////////////////////////////////////////////////////////////////
//print results in a database format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, int iNumDevs, bool wc)
{
    std::string sConfig;

    // log config information
    if (kind == DEVICE_TO_DEVICE)
    {
        sConfig += "D2D";
    }
    else
    {
        if (kind == DEVICE_TO_HOST)
        {
            sConfig += "D2H";
        }
        else if (kind == HOST_TO_DEVICE)
        {
            sConfig += "H2D";
        }

        if (memMode == PAGEABLE)
        {
            sConfig += "-Paged";
        }
        else if (memMode == PINNED)
        {
            sConfig += "-Pinned";

            if (wc)
            {
                sConfig += "-WriteCombined";
            }
        }
    }

    unsigned int i;
    double dSeconds = 0.0;

    for (i = 0; i < count; i++)
    {
        dSeconds = (double)memSizes[i] / (bandwidths[i] * (double)(1<<20));
        printf("bandwidthTest-%s, Bandwidth = %.1f MB/s, Time = %.5f s, Size = %u bytes, NumDevsUsed = %d\n",
               sConfig.c_str(), bandwidths[i], dSeconds, memSizes[i], iNumDevs);
    }
}



__global__ void delay(volatile int *flag, unsigned long long timeout_clocks = 100000000)
{
    // Wait until the application notifies us that it has completed queuing up the
    // experiment, or timeout and exit, allowing the application to make progress
    long long int start_clock, sample_clock;
    start_clock = clock64();

    while (!*flag) {
        sample_clock = clock64();

        if (sample_clock - start_clock > timeout_clocks) {
            break;
        }
    }
}

void performHostToDeviceCopy(int *dest, int *src, int num_elems, int repeat, hipStream_t streamToRun)
{
	for (int r = 0; r < repeat; r++) {
		checkCudaErrors(hipMemcpyAsync(dest, src, sizeof(int)*num_elems, hipMemcpyHostToDevice, streamToRun));
	}
}

void performDeviceToHostCopy(int *dest, int *src, int num_elems, int repeat, hipStream_t streamToRun)
{
	for (int r = 0; r < repeat; r++) {
		checkCudaErrors(hipMemcpyAsync(dest, src, sizeof(int)*num_elems, hipMemcpyDeviceToHost, streamToRun));
	}
}

void performDeviceToDeviceCopy(int *dest, int *src, int num_elems, int repeat, hipStream_t streamToRun, int deviceID)
{
	for (int r = 0; r < repeat; r++) {
		checkCudaErrors(hipMemcpyAsync(dest, src, sizeof(int)*num_elems, hipMemcpyDeviceToDevice, streamToRun));

	}
}


void outputLatencyMatrix(int numGPUs, bool p2p, memcpyKind kind)
{
    int repeat = MEMCOPY_ITERATIONS_For_Test_Latency;
    volatile int *flag = NULL;
    StopWatchInterface *stopWatch = NULL;
    std::vector<int *> buffersH(numGPUs);  // buffer for D2D, that is, intra-GPU copy
    std::vector<int *> buffersD(numGPUs);
    std::vector<int *> buffersD2D(numGPUs);
    std::vector<hipStream_t> stream(numGPUs);
    std::vector<hipEvent_t> start(numGPUs);
    std::vector<hipEvent_t> stop(numGPUs);

    hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
    cudaCheckError();

    if (!sdkCreateTimer(&stopWatch)) {
        printf("Failed to create stop watch\n");
        exit(EXIT_FAILURE);
    }
    sdkStartTimer(&stopWatch);

    for (int d = 0; d < numGPUs; d++) {
        hipSetDevice(d);
        hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
        hipHostAlloc((void **)&buffersH[d], sizeof(int), hipHostMallocPortable);
        hipMalloc(&buffersD[d], sizeof(int));
        hipMalloc(&buffersD2D[d], sizeof(int));
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    std::vector<double> gpuLatencyMatrix(numGPUs);
    std::vector<double> cpuLatencyMatrix(numGPUs);

    for (int i = 0; i < numGPUs; i++) {
        hipSetDevice(i);
        hipStreamSynchronize(stream[i]);
        cudaCheckError();

        // Block the stream until all the work is queued up
        // DANGER! - hipMemcpy*Async may infinitely block waiting for
        // room to push the operation, so keep the number of repeatitions
        // relatively low.  Higher repeatitions will cause the delay kernel
        // to timeout and lead to unstable results.
        *flag = 0;
        delay<<< 1, 1, 0, stream[i]>>>(flag);
        cudaCheckError();
        hipEventRecord(start[i], stream[i]);

        sdkResetTimer(&stopWatch);

        if (kind == HOST_TO_DEVICE)
        {
            performHostToDeviceCopy(buffersD[i], buffersH[i], 1, repeat, stream[i]);
        }
        else if (kind == DEVICE_TO_HOST)
        {
            performDeviceToHostCopy(buffersH[i], buffersD[i], 1, repeat, stream[i]);
        }
        else if (kind == DEVICE_TO_DEVICE)
        {
        	performDeviceToDeviceCopy(buffersD2D[i], buffersD[i], 1, repeat, stream[i], i);
        }

        hipEventRecord(stop[i], stream[i]);
        // Now that the work has been queued up, release the stream
        *flag = 1;
        hipStreamSynchronize(stream[i]);
        cudaCheckError();

        float cpu_time_ms = sdkGetTimerValue(&stopWatch);

        float gpu_time_ms;
        hipEventElapsedTime(&gpu_time_ms, start[i], stop[i]);

        gpuLatencyMatrix[i] = (double)gpu_time_ms * 1e3 / repeat;
        cpuLatencyMatrix[i] = (double)cpu_time_ms * 1e3 / repeat;

    }

    printf("  GPU side latency\n");

    for (int i = 0; i < numGPUs; i++) {
        printf("%8d ", i);
    }
    printf("\n");

    for (int i = 0; i < numGPUs; i++) {
        printf("%8.05f ", gpuLatencyMatrix[i]);
    }
    printf("\n");

    printf("  CPU side latency\n");

    for (int i = 0; i < numGPUs; i++) {
        printf("%8d ", i);
    }
    printf("\n");

    for (int i = 0; i < numGPUs; i++) {
        printf("%8.05f ", cpuLatencyMatrix[i]);
    }
    printf("\n");

    for (int d = 0; d < numGPUs; d++) {
        hipSetDevice(d);
        cudaCheckError();
        hipHostFree(buffersH[d]);
        cudaCheckError();
        hipFree(buffersD[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
        hipStreamDestroy(stream[d]);
        cudaCheckError();
    }

    sdkDeleteTimer(&stopWatch);

    hipHostFree((void *)flag);
    cudaCheckError();
}


///////////////////////////////////////////////////////////////////////////
//Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void)
{
    printf("Usage:  bandwidthTest [OPTION]...\n");
    printf("Test the bandwidth for device to host, host to device, and device to device transfers\n");
    printf("\n");
    printf("Example:  measure the bandwidth of device to host pinned memory copies in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments\n");
    printf("./bandwidthTest --memory=pinned --mode=range --start=1024 --end=102400 --increment=1024 --dtoh\n");

    printf("\n");
    printf("Options:\n");
    printf("--help\tDisplay this help menu\n");
    printf("--csv\tPrint results as a CSV\n");
    printf("--device=[deviceno]\tSpecify the device device to be used\n");
    printf("  all - compute cumulative bandwidth on all the devices\n");
    printf("  0,1,2,...,n - Specify any particular device to be used\n");
    printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
    printf("  pageable - pageable memory\n");
    printf("  pinned   - non-pageable system memory\n");
    printf("--mode=[MODE]\tSpecify the mode to use\n");
    printf("  quick - performs a quick measurement\n");
    printf("  range - measures a user-specified range of values\n");
    printf("  shmoo - performs an intense shmoo of a large range of values\n");

    printf("--htod\tMeasure host to device transfers\n");
    printf("--dtoh\tMeasure device to host transfers\n");
    printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
    printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
    printf("--cputiming\tForce CPU-based timing always\n");

    printf("Range mode options\n");
    printf("--start=[SIZE]\tStarting transfer size in bytes\n");
    printf("--end=[SIZE]\tEnding transfer size in bytes\n");
    printf("--increment=[SIZE]\tIncrement size in bytes\n");
}
